#include "hip/hip_runtime.h"
#include "state.cuh"
#include <cstdint>

state::state()
{

}

double state::getPriorty(int index, int expSize)
{
	return ((index / expSize)/totalEXP);
}

double state::getTrueValue()
{
	return value / visited;
}

bool state::setValue(double newValue)
{
	this->value = newValue;
	return true;

}

double state::getValue()
{
	return value;
}

bool state::safeSet(double newValue)
{
	double newVal = newValue;
	double tempValue = value; 
	uint64_t old = atomicCAS((uint64_t*)&value,*((uint64_t*)&tempValue), *((uint64_t*)&newVal));
	double* convertDouble = (double*) &old;
	float*  convert = (float*) & old;
	printf("\n THIS IS THE VALUE %f", value);
	
	
	if (*convertDouble == tempValue)
	{
		return true;
	}
	else
	{
		return false; 
	}
}

int state::getVisited()
{
	return visited;
}

void state::setVisited(int newVisitCount)
{
	visited = newVisitCount;
}

void state::setOldValue()
{
	oldValue = value;
}



