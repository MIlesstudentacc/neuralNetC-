#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "vector"
#include "state.cuh"
#include "thrust/device_vector.h"
#include "device_atomic_functions.h"
#include <hip/hip_runtime.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

//need to imeplent the min condition 

__device__ double globalDouble = 0;

__device__ int test = 50;

__device__ volatile int incrementor = 0;

__device__ int* arr[5];

__device__  double gpu = 5;

__device__ int Sum[7] = { 1,2,3,4,5,6,7 };

__device__ int len = 7;



__device__ int atomicJobGrab = 0;

__device__ int atomicJobGrabShift = 0;

__device__ volatile int globalFound = -1; 

__device__ state* saveState;



__device__ state* EXPStore[100];

__device__ state** EXP = EXPStore;

__device__ state* EXPReplaceStore[100];

__device__ state** EXPReplace = EXPReplaceStore;

__device__ int atomicSynch = 0; //may need to atomicCAS this instead volatile

__device__ int deviceMin =0; 

__device__ double lockValue = -1;

__device__ double threadLock = -1;

#define MAX RAND_MAX

__global__ void addToExp(unsigned int seed)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    state* newState = new state(); 
    hiprandState_t curState;
    hiprand_init(seed, index, 0, &curState);
    for (int i = index; i < 100; i = i + stride)
    {
        state* newState = new state();
        double newValue = i;
        newState->setValue(newValue);
        EXP[i] = newState;
    }

}

/*
void readEXP()
{
    for (int i = 0; i < 100; i++)
    {
        std::cout << EXP[i]->getValue() << std::endl; 
    }
}
void addToExp()
{
    state* newState = new state();
    for (int i = 0; i < 100;i++)
    {
        state* newState = new state();
        double newValue = i;
        newState->setValue(newValue);
        EXP[i] = newState;
    }


}*/
__host__
void simpleBinarySearch(unsigned int seed)
{
    //int index = blockIdx.x * blockDim.x + threadIdx.x;
    //int stride = blockDim.x * gridDim.x;
    //hiprandState_t curState;
    //hiprand_init(seed, index, 0, &curState);
    
    int startIndex = 50;
    int diff;
    state* newState = new state();
    double newValue = (double)fmod((double)rand(), (double)1000);

    newState->setValue(newValue);
    state* compare;
    state* compareNext;
    int boundaryIndex = 100;
    bool key = false;
    
    for (int i = 0; i < 100; i = i ++)
    {
        
        startIndex = 50;
       


        newValue = (double)fmod((double)rand(), (double)1000);

        newState->setValue(newValue);

        boundaryIndex = 100;
        key = false;
       
        if (newState->getValue() > EXP[startIndex]->getValue())
        {

            diff = sqrt(pow(startIndex - boundaryIndex, 2)) / 2;
            startIndex = startIndex + diff;

        }
        if (newState->getValue() <= EXP[startIndex]->getValue())
        {
            diff = sqrt(pow(startIndex - boundaryIndex, 2)) / 2;
            startIndex = startIndex - diff;
        }
        int counter = 0; 

        while (key == false)
        {
            counter = counter + 1;
            compare = nullptr;
            compareNext = nullptr;
            printf("start index %d", startIndex);
            compare = EXP[startIndex];
            if (startIndex < 99)
            {
                compareNext = EXP[startIndex + 1];
                printf("\n compare state next %f", compareNext->getValue());
            }
            printf("new state %f \n ", newState->getValue());
            printf("compare state %f \n ", compare->getValue());

            //printf("compare next state %f \n ", compareNext->getValue());
            if (newState->getValue() > compare->getValue())
            {
                if (diff > 1)
                {
                    diff = diff / 2;
                }
                startIndex = startIndex + diff;
            }
            if (newState->getValue() <= compare->getValue())
            {
                if (diff > 1)
                {
                    diff = diff / 2;
                }
                startIndex = startIndex - diff;
            }

            double compareVala = compare->getValue();
            double newStateValue = newState->getValue();

            if (newStateValue > compareVala)
            {
                
                if (compareNext==nullptr)
                {
                    printf("runs pst 1st if");
                    key = true;
                }
                else
                {
                    double compareValb = compareNext->getValue();

                    if (newStateValue <= compareValb)
                    {
                        //printf("compare done \n");
                        key = true;
                    }
                    else
                    {
                        //printf("THIS STATEMENT IS NOT FUCKING TRUE");
                    }
                }
            }
        }

        key = false;

        while (key == false)
        {

            bool Race = compare->setValue(newState->getValue());
            if (Race)
            {
                key = true;
                /*free(newState);
                free(compare);
                if (compareNext != nullptr)
                {
                    free(compareNext);
                }*/
            }
            else
            {
                startIndex = startIndex - 1;
                compare = EXP[startIndex];
            }
        }
        if (startIndex > 0)
        {
            EXP[startIndex] = compare;
            printf("\n %d done \n", i);
        }
        else
        {
            printf("\n - hit\n");
        }
    }

}
__global__
void displayEXP()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    printf("is this even running");
    for (int i = index; i < 100; i = i + stride)
    {
        printf("%f \n", EXP[i]->getValue());
    }
      
}
/*
__global__ void binarySearch(unsigned int seed)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    hiprandState_t curState;
    hiprand_init(seed, index, 0, &curState);
    bool key = false; 
    state* comparePointer; 
    for (int i = index; i < 500; i++)
    {
        state* newState = new state(); 
        double newValue = (double)fmod((double)hiprand(&curState), (double)100);
        newState->setValue(newValue);
        int lastIndex;
        int startIndex = 50;
        int lenEXP = 100;

        bool greaterOrLesser;
        if (newState->getValue() > comparePointer->getValue())
        {
            startIndex = startIndex + ((lenEXP - startIndex)/2);
            greaterOrLesser = true; 
            while (greaterOrLesser == true)
            {
                comparePointer = EXP[startIndex];
                if (newState->getValue() > comparePointer->getValue())
                {
                    lastIndex = startIndex;
                    startIndex = startIndex + ((lenEXP - lastIndex)/2);

                }
                if (newState->getValue() < comparePointer->getValue())
                {
                    greaterOrLesser = false;
                
                }
            }
            while (key == false)
            {
                int addOn = 1; 
                lastIndex = lastIndex + addOn; 
                
                if (lastIndex < 0)
                {
                    break;
                }
                comparePointer = EXP[lastIndex];
                if (newState->getValue() > comparePointer->getValue())
                {
                    
                    
                    bool Race = comparePointer->safeSet(newState->getValue());
                    if (Race)
                    {
                        key = true;

                    }
                    else
                    {
                        addOn = -1;
                    }
                    
                }

            }
        }
       //reversed operation
        if (newState->getValue() < comparePointer->getValue())
        {
            startIndex = startIndex / 2;
            greaterOrLesser = false;
            while (greaterOrLesser == false)
            {
                comparePointer = EXP[startIndex];
                if (newState->getValue() < comparePointer->getValue())
                {
                    lastIndex = startIndex;
                    startIndex = startIndex/2;

                }
                if (newState->getValue() > comparePointer->getValue())
                {
                    greaterOrLesser = true;


                }
            }
            while (key == false)
            {
                int addOn = 1;
                lastIndex = lastIndex + addOn;
                comparePointer = EXP[lastIndex];

                if (newState->getValue() > comparePointer->getValue())
                {
                    
                    bool Race = comparePointer->safeSet(newState->getValue());
                    if (Race)
                    {
                        key = true;

                    }
                    else
                    {
                        addOn = -1;
                    }
                    
                }

            }
        }
    }
    
}
*/
__global__ void hitAndShift(unsigned int seed)
{
    
    hiprandState_t curState;
    hiprand_init(seed, 0, 0, &curState);
    double range = (double)fmod((double)hiprand(&curState), (double)MAX);
    int lenExp = 100;
    int total = (100 * 99) / 2 / 100;
    double r = range;
    //
    // Coefficients for the quadratic equation : ax ^ 2 + bx + c = 0
    int a = 1;
    int b = 1;
    double c = -2 * r * lenExp * total;

    //Solving the quadratic equation for n using the quadratic formula
    double discriminant = sqrt(pow(b, 2) - 4 * a * c);
    double n1 = (-b + discriminant) / (2 * a);
    double n2 = (-b - discriminant) / (2 * a);

    //Selecting the positive root since n must be positive]
    int n;
    if (n1 > 0)
    {
        n = n1;
    }
    else
    {
        n = n2;
    }

    //Rounding down
    int id = n;//(Daniel,Kramer,2024)

    //state* compare = EXP[id];
    //compare->setVisited(compare->getVisited());
    //compare->getTrueValue();

    

}



__global__ void testAtomicCas()

{
    double compare = 0; 
    double newValue = 5; 

    uint64_t old = atomicCAS((uint64_t*)&globalDouble, *((uint64_t*)&compare), *((uint64_t*)&newValue));

    printf("%f", globalDouble);


}


__global__ void observableState()
{
   
   
  
    int id = threadIdx.x;
    printf("yello %d \n",incrementor);
    while (id != incrementor)
    {
        printf("var %d \n", incrementor);
        //atomicCAS(&incrementor, id, 1);
       
    }
    //atomicAdd(&incrementor, 1);
    incrementor = incrementor + 1;
    printf("hello world!"); 
    
}
__global__ void randomK(unsigned int seed)
{
    double gpu2;
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);
    for (int i = 0; i < 10; i++)
    {
        gpu2 = (double) fmod((double) hiprand(&state),(double)MAX);
        
        gpu2 = gpu2/MAX;
        printf("%f \n", gpu2);
    
    }
    
}

__global__ 
void runIn()
{
  
    printf("hello");
    __shared__ int s[64];
    s[1] = 5;
    //__syncthreads();
    printf("value");
    
}
__global__
void runShare()
{
    printf("hello");
    extern __shared__ int s[];
    //__syncthreads();
 
    while (s[1] != 5)
    {

    }
    printf("rar");
    printf("%d", s[1]);
}

__global__
void display()
{
    
   // thrust::device_vector<int> myvec;
    //myvec.push_back(1);
    printf("hello ther");
}

__global__ void atomAdd(int *a_d)
{
    //atomicAdd(&incrementor, 1);
    printf("%d", incrementor);
    
}

__global__ void freshKernel()
{
    int threadID = threadIdx.x;

    for (int i = 0; i < 5; i++)
    {
        if (threadIdx.x == incrementor)
        {
            incrementor = incrementor + 1;
       
        }
        __syncthreads();
    }
    printf("%d", incrementor);
}
__global__  
void raceAdd(int* testValP)
{
    
    
    __shared__ int Var1;
    
    printf("%d", Var1);

    printf("%d", test);

    int threadUpdate = *testValP;
    printf("thread updates 1 %d \n", threadUpdate);
    int threadID = threadIdx.x;
    //printf("%d this is the id", threadIdx.x);
    if (threadIdx.x == 0)
    {
        Var1 = 1;
        printf("if passed \n ");
    }
    printf("do changes occur %d",Var1);

    printf("%d \n", threadIdx.x);
    printf("hello hello hello \n");
    printf("thread updates 2 %d \n", threadUpdate);
    while (threadID != threadUpdate)
    {
        threadUpdate = *testValP;
    }
    *testValP = *testValP + 1;
    printf("loop passed \n");
 
}

__device__
int simpleBinarySearchPortion(int startIndex,int jobSize,double value,bool appendEnd)
{
    int numChecked = 0; 
    state* newState = new state();
    int originalIndex = startIndex;
    newState->setValue(value);
    state* compare;
    state* compareNext;
    int boundaryIndex = jobSize;
    int diff = (boundaryIndex-startIndex)/2;
    if (diff <= 0)
    {
        diff = 1;
    }
    bool key = false;
    while (key == false)
    {
        if ((startIndex < originalIndex || globalFound > -1) && lockValue>-1)
        {
            break;
        }
        compareNext = nullptr;

        compare = EXP[startIndex];
        if (startIndex < jobSize )
        {
            compareNext = EXP[startIndex + 1];
        }
        if (newState->getValue() > compare->getValue())
        {
            if (diff > 1)
            {
                diff = diff / 2;
            }
            
            if ((compareNext == nullptr && appendEnd)|| (compareNext->getValue() >= newState->getValue()))
            {
                printf("hello");
                key = true; 
            }
            else
            {
                startIndex = startIndex + diff;
            }

        }
        if (newState->getValue() <= compare->getValue())
        {
            if (diff > 1)
            {
                diff = diff / 2;
            }
            startIndex = startIndex - diff;
            
        }
        numChecked = numChecked+1;
        if (numChecked >= jobSize-originalIndex)
        {
            break;
        }
    }
    if (key)
    {
        saveState = new state();
        saveState->setValue(EXP[startIndex]->getValue());
        
        EXP[startIndex]->setValue(newState->getValue());
        return startIndex;
    }
    else
    {
        return -1; 
    }

}


__device__
void teamBinarySearch(int jobSize, double value)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    while(true)//value >-1
    { 
        int miniJobSize = (jobSize / stride);
        if (jobSize % stride > 0)
        {
            miniJobSize = miniJobSize + 1;
        }
        int old = atomicAdd(&atomicJobGrab,1);

        int startIndex = (miniJobSize * old);
        if ((startIndex+miniJobSize-1) < jobSize)
        {
            bool appendEnd = false;
            if (startIndex + miniJobSize == jobSize)
            {
                //printf("start index %d", startIndex);
                appendEnd = true;

            }
            int result = simpleBinarySearchPortion(startIndex, miniJobSize + startIndex, value, appendEnd);
            if (result >-1)
            {
                atomicJobGrab = 0;
                if (result == 0)
                {
                    int deviceMin = result;
                }
                globalFound = result;
                
                
            }
            if (globalFound >-1 || lockValue ==-1)
            {
                break;
            
            }
        }
       
    //this is where shifting will be done

    }
}

__global__ 
void copyToReplace()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < 100; i = i + stride)
    {
        EXPReplace[i] = EXP[i];
    }
}

__device__
void shiftPortion(int startIndex, int jobSize)
{
    while (startIndex < jobSize)
    {
        if (startIndex == 0)
        {
            free(EXPReplace[startIndex]);
        }
        EXPReplace[startIndex] = EXP[startIndex + 1];
        startIndex = startIndex + 1;

    }
    atomicAdd(&atomicSynch, 1);

}

__device__
void kernalShift(int jobSize,int value)
{
    jobSize = globalFound;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int miniJobSize = (jobSize / stride);
    if (jobSize % stride > 0)
    {
        miniJobSize = miniJobSize + 1;
    }
    while (atomicSynch < jobSize / miniJobSize && globalFound>-1)
    {
        int old = atomicAdd(&atomicJobGrabShift, 1);
       
        
        printf("\n %d %d %d", old, atomicSynch, jobSize / miniJobSize);
        int startIndex = (miniJobSize * old);
        if ((startIndex + miniJobSize-1) < jobSize)
        {
            shiftPortion(startIndex, miniJobSize + startIndex);
        }
     
    }
    if (atomicSynch == jobSize / miniJobSize || value ==-1)
    {
        printf("%d",atomicSynch);
        EXPReplace[globalFound-1] = saveState;
        EXP = EXPReplace;
        

    }


}
__global__
void setSaveState()
{
    saveState = new state();
  

}
__device__
void checkLock(int jobsize, double value)
{
    if (threadLock == -1)
    {
        lockValue = value;//this will be a random value not 5
        atomicJobGrab = 0; 
        threadLock = threadIdx.x;
        globalFound = -1;
        atomicJobGrabShift = 0; 
        atomicSynch = 0;



    }
    if (lockValue > -1)
    {
        //do all the complex shit
        teamBinarySearch(jobsize, value);
        //lock already done
        kernalShift(globalFound-1,value);

        if (threadLock == threadIdx.x)
        {
            lockValue = -1;
           


        }
    }
}
__global__
void generate(int jobSize,unsigned int seed)
{
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);
    for (int i = 0; i < 1; i++)
    {
        double localValue = (double)fmod((double)hiprand(&state), (double)100);

        if (localValue > deviceMin)
        {
            checkLock(jobSize, localValue);
        }
        printf("hopefuly this works \n%f", localValue);
    }
}

int main() 
{   
    srand(time(0));
    /*
    int a = 0, * a_d;
    hipMalloc((void**)&a_d,sizeof(int));
    hipMemcpy(a_d, &a, sizeof(int),hipMemcpyHostToDevice);

    atomAdd << <1, 5 >> > (a_d);
    */
    unsigned int seed = time(NULL); 
    //testAtomicCas << <1, 1 >> > ();
    //addToExp();

    //readEXP();
    addToExp<<<1, 1>>>(seed);
    int jobsize = 100;
    copyToReplace << <1, 50 >> > ();
    
    generate<<<1, 50>>>(100,time(NULL));
    displayEXP<<<1, 1>>>();

}
