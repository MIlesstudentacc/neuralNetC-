#include "hip/hip_runtime.h"
#include "node.cuh"

void node::setWeightsSize(int weightSize)
{
	weights = new double[weightSize];
}

void node::updateWeight(int weightID, double weight)
{
	weights[weightID] = weight;
}

void node::updateBias(double bias)
{
	this->bias = bias; 
}


double node::getActivation()
{
	return activation;
}

double node::getWeight(int weightID)
{
	return weights[weightID];
}

void node::setActivation(double newActivation)
{
	activation = newActivation;
}

void node::IncrementMergePopulation()
{
	atomicAdd(&populateCount, 1);
}

void node::addToRawForwardSum(double weightActivateMerge)
{
	atomicAdd(&rawForwardSum, weightActivateMerge);
}

int node::returnPopulationCount()
{
	return populateCount;
}

double node::getRawForwardSum()
{
	return rawForwardSum;
}

double node::getBias()
{
	return bias;
}

bool node::getActivated()
{
	return true;
}

void node::setActivated(bool activated)
{
	this->activated = activated;
}

void node::setWeightUpdateBatch(int batch)
{
	weightUpdates = new double*[batch];
	for (int i = 0; i < batch; i++)
	{
		weightUpdates[i] = new double[batch];
	}
	

}

void node::updateWeightUpdate(double weight, int weightID, int batch)
{
	weightUpdates[weightID][batch] = weight; 
}

void node::setBiasUpdateBatch(int batch)
{
	biasUpdates = new double[batch];
}

void node::updateBiasUpdates(int batch,double newBias)
{
	biasUpdates[batch] = newBias;
}



