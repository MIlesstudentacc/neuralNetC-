#include "hip/hip_runtime.h"
#include "node.cuh"

void node::setWeightsSize(int weightSize)
{
	weights = new double[weightSize];
}

void node::updateWeight(int weightID, double weight)
{
	weights[weightID] = weight;
}

void node::updateBias(double bias)
{
	this->bias = bias; 
}


double node::getActivation()
{
	return activation;
}

double node::getWeight(int weightID)
{
	return weights[weightID];
}

void node::setActivation(double newActivation)
{
	activation = newActivation;
}

void node::IncrementMergePopulation()
{
	atomicAdd(&populateCount, 1);
}

void node::addToRawForwardSum(double weightActivateMerge)
{
	atomicAdd(&rawForwardSum, weightActivateMerge);
}

int node::returnPopulationCount()
{
	return populateCount;
}

double node::getRawForwardSum()
{
	return rawForwardSum;
}

double node::getBias()
{
	return bias;
}

bool node::getActivated()
{
	return true;
}

void node::setActivated(bool activated)
{
	this->activated = activated;
}

void node::setWeightUpdateBatch(int batch)
{
	weightUpdates = new double*[batch];
	for (int i = 0; i < batch; i++)
	{
		weightUpdates[i] = new double[batch];
	}
	

}

void node::updateWeightUpdate(double weight, int weightID, int batch)
{
	weightUpdates[weightID][batch] = weight; 
}

void node::setBiasUpdateBatch(int batch)
{
	biasUpdates = new double[batch];
}

void node::updateBiasUpdates(int batch,double newBias)
{
	biasUpdates[batch] = newBias;
}

void node::updateWeights(int batchSize,int nextNeuronCount,double LEARNING_RATE)
{
	double averageWeight = 0;
	for (int weightID = 0; weightID < nextNeuronCount; weightID = weightID + 1)
	{
		for (int batchID = 0; batchID < batchSize; batchID++)
		{
			averageWeight = averageWeight + weightUpdates[batchID][weightID];
		}
		averageWeight = averageWeight / batchSize;
		double old_weight = getWeight(weightID);
		double new_weight = old_weight - (averageWeight * LEARNING_RATE);
		weights[weightID] = new_weight;
	}
}

void node::updateBiases(int batchSize, double LEARNING_RATE)
{
	double averageBias = 0; 
	for (int batchID = 0; batchID < batchSize; batchID++)
	{
		averageBias = averageBias + biasUpdates[batchID];
	}
	double old_bias = bias;
	double new_bias = bias - (averageBias * LEARNING_RATE);
	bias = new_bias;
}



