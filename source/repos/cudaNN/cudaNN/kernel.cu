#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "network.cuh"
#include <iostream>


__global__
void testSig()
{
	
	
}


__global__
void newIt()
{
	node* myNode = new node();
	printf("%f", myNode->getActivation());
	myNode->setActivation(5);
	printf("%f", myNode->getActivation());
}
__global__ 
void lazyMemoryPopulate(network* myNetwork)
{
	//printf("%d", myNetwork->totalLayers);
	myNetwork->addNeuronObjects();
}
__global__
void generateWeights(network* myNetwork,unsigned int seed)
{

	myNetwork->populateNetworkStructFixed(seed);

}
__global__
void forwardProp(network* myNetwork,int layer)
{
	
	myNetwork->forwardPropagate(layer);
	printf("finished forward prop");
}
__global__
void displayKernelAct(network* myNetwork)
{
	printf("hello world");
	myNetwork->displayActivations();
}
__global__
void setInputs(network* myNetwork,double* inputs)
{
	myNetwork->setInputs(inputs);
}


int main()
{
	//network* mynetwork = new network();
	//mynetwork->sigmoid(5);
	
	network* myNetwork;

	network* myNetworkCPU = new network();


	hipMalloc(&myNetwork, sizeof(network));
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	hipMemcpy(myNetwork, myNetworkCPU, sizeof(network), hipMemcpyHostToDevice);

	int total = myNetworkCPU->totalLayers;
	lazyMemoryPopulate << <1, 1 >> > (myNetwork);
	generateWeights << <1, 1>> > (myNetwork, time(NULL));
	//no inputs yet muppet
	double myList[5] = {0.54,0.32,0.23,0.74,0.82};

	setInputs << <1, 1 >> > (myNetwork, &myList);
	forwardProp << <1,1 >> > (myNetwork,0);
	

	hipDeviceSynchronize();
	displayKernelAct << <1, 1 >> > (myNetwork);
	hipDeviceSynchronize();
	std::cout << "all done" << std::endl;

	
}