#include "hip/hip_runtime.h"
#include "network.cuh"
#include <iostream>



 
network::network()
{
    //cutting out for ease of testing

    std::cout << "how many layers will be hidden layers" << std::endl;
    std::cin >> hiddenLayers;
    neuronCounts = new int[hiddenLayers + 2];
    std::cout << "how many neurons will be in the input layer" << std::endl;
    std::cin >> inputNeurons;
    neuronCounts[0] = inputNeurons;
   
    for (int i = 1; i < hiddenLayers+1; i++)
    {
        std::cout << "how many neurons will be in the hidden layers" << std::endl;
        std::cin >> hiddenNeurons;
        neuronCounts[i] = hiddenNeurons;
    }
    std::cout << "how many neurons will be in the output layer" << std::endl;
    std::cin >> outputNeurons;
    neuronCounts[hiddenLayers+1] = outputNeurons;

    /*
    inputNeurons = 5;
    hiddenNeurons = 8;
    hiddenLayers = 3;
    outputNeurons = 5;*/
    totalLayers = hiddenLayers + 2;
    int* neuronCountsCopy = neuronCounts;
    hipMalloc(&neuronCounts, totalLayers * sizeof(int));
    hipMemcpy(neuronCounts, neuronCountsCopy, totalLayers * sizeof(int), hipMemcpyHostToDevice);
    ;
    desiredOutcome = new double[outputNeurons];
    double* desiredOutcomeCopy = desiredOutcome;
    hipMalloc(&desiredOutcome, outputNeurons * sizeof(double));
    hipMemcpy(desiredOutcome, desiredOutcomeCopy, outputNeurons * sizeof(double), hipMemcpyHostToDevice);
    taskCount = new int[totalLayers];
    for (int i = 0; i < totalLayers-1; i++)
    {
        taskCount[i] = neuronCountsCopy[i] * neuronCountsCopy[i + 1];
        taskCount[i] = taskCount[i] + neuronCountsCopy[i + 1];
        totalTaskCount = totalTaskCount + taskCount[i];
    }
    int* taskCountCopy = taskCount;
    hipMalloc(&taskCount, totalLayers * sizeof(int));
    hipMemcpy(taskCount, taskCountCopy, totalLayers * sizeof(int), hipMemcpyHostToDevice);

}

void network::addNeuronObjects()
{
    networkStructure = new node **[totalLayers];
    for (int layer = 0; layer < totalLayers; layer++)
    {
        node** localList = new node*[neuronCounts[layer]];

        for (int nodeID = 0; nodeID < neuronCounts[layer]; nodeID++)
        {
            node* newNode = new node();
            newNode->setWeightsSize(neuronCounts[layer + 1]);
            localList[nodeID] = newNode;
            
        }

        networkStructure[layer] = localList;

    }
}


double network::sigmoid(double x)
{
	
	double result;
	result = 1 / (1 + exp(-x));
	return result;
}

void network::populateNetworkStructFixed(unsigned int seed)
{


    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int offset = 0;
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);
    int total = totalTaskCount;
    int count = 0;
    
    double weight;
    for (int taskID = index; taskID < totalTaskCount; taskID = taskID + stride)
    {
        //unsigned int seed = time(NULL);
        count = 0;
        offset = 0;
        for (int layer = 0; layer < totalLayers; layer = layer +1)
        {
            count = count + taskCount[layer];
            if (taskID < count)
            {
                int localID = taskID - offset;
                int neuron = (localID / neuronCounts[layer+1]);
                int weightID = localID % neuronCounts[layer + 1];
                if (neuron > (neuronCounts[layer]-1))
                {
                    
                    node* nodeUpdate = networkStructure[layer + 1][weightID];
                    generateBias(nodeUpdate, weightID,layer ,state);
                }
                else
                {
                    
                    node* nodeUpdate = networkStructure[layer][neuron];
                    generateWeights(nodeUpdate, weightID, layer,state);
                }
                offset = offset + taskCount[layer];
                break;

            }
            offset = offset + taskCount[layer];
        }
    }

}

void network::generateWeights(node* toGetWeight,int weightID,int layers, hiprandState_t &state)
{

    double weight;
    int weightCount = neuronCounts[layers + 1];

    double range = sqrt((float) weightCount);
    double MAX = RAND_MAX;
   // for (int i = 0; i < 10; i++)
   // {
         weight = (double)fmod((double)hiprand(&state), (double)MAX);
   
   // }
    weight = weight / MAX;
    printf("pre-range set %f             ", weight);
    weight = (double) fmod((double) weight, (double) range);
    printf("%f             \n", weight);
   
    int flip =  hiprand(&state)%2;
    if (flip)
    {
        weight = weight * -1;
    }
    toGetWeight->updateWeight(weightID,weight);
}

void network::generateBias(node* toGetWeight, int weightID, int layers, hiprandState_t &state)
{
   
    int weightCount = neuronCounts[layers + 1];
    double range = sqrt((float) weightCount);
    int MAX = RAND_MAX;
    double bias = (double)fmod((double)hiprand(&state), (double)MAX);
    bias = bias / MAX;
    bias = (double)fmod((double)bias, (double)range);
    int flip = hiprand(&state) % 2;
    if (flip)
    {
        bias = bias * -1;
    }
    toGetWeight->updateBias(bias);
}

void network::forwardPropagate(int layer)
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

  
    int weightID;
    int neuron;
    node* targetObj;
   
    
    for (int taskID = index; taskID < taskCount[layer]; taskID = taskID + stride)
    {
        neuron = taskID / neuronCounts[layer] ;
        
        weightID = taskID % neuronCounts[layer+1];
        node* neuronObj = networkStructure[layer][neuron];
        targetObj = networkStructure[layer + 1][weightID];
        if (weightID ==0)
        {
            neuronObj->setActivated(sigmoid(neuronObj->getActivation()));
        }
        if (neuron < neuronCounts[layer])
        {
            targetObj->addToRawForwardSum(neuronObj->getActivation() * neuronObj->getWeight(weightID));
        }
        else
        {
            targetObj->addToRawForwardSum(neuronObj->getActivation()*neuronObj->getBias());
        }


    }
      
    
    
}

void network::activateNeuron(int size,node* neuron,int layer)
{
  
    if (neuron->returnPopulationCount() == size)
    {
        if (layer < totalLayers - 2)
        {
            neuron->setActivation(sigmoid(neuron->getRawForwardSum()));
            neuron->setActivated(true);
        }
        else
        {
            neuron->setActivation(neuron->getRawForwardSum());
        }
    }
    

}

void network::displayActivations()
{
    int total = totalLayers;
    for (int layers = 0; layers < total; layers = layers+1)
    {
        int count = neuronCounts[layers];
        printf("____________________________________________\n");
        for (int countID = 0; countID < count; countID = countID + 1)
        {
            printf("%f              ", networkStructure[layers][countID]->getActivation());
        }

    }
}

void network::setDesiredOutcome(int pos, double replace)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int outcomeID = index; outcomeID < neuronCounts[totalLayers - 1]; outcomeID = outcomeID + stride)
    {
        
        desiredOutcome[outcomeID] = 0;
        if (outcomeID == pos)
        {
            desiredOutcome[outcomeID] = replace; 
        }
    }
    
}

void network::calcDefaultDeriv()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double derivCost;
    if (threadIdx.x == 0)
    {
        deriv = new double[totalLayers];
    }
    __syncthreads(); 
    for (int nodeID = index; nodeID < neuronCounts[totalLayers - 1]; index + stride)
    {
        node* neuron = networkStructure[totalLayers - 1][nodeID];
        if (classification == true)
        {
            double allExp = allExpOutput();
            derivCost = derivativecost(softMaxSingle(allExp, nodeID), desiredOutcome[nodeID]);
        }
        else
        {
            derivCost = derivativecost(neuron->getActivation(), desiredOutcome[nodeID]);
        }
        deriv[nodeID] = derivCost;
    }
}

double network::derivativecost(double activation, double desiredOutcome)
{
    return(2 * (activation - desiredOutcome));
}

double network::softMaxSingle(double allExp, int numeriator)
{
    return exp(networkStructure[totalLayers - 1][numeriator]->getActivation()) / allExp;
}

double network::allExpOutput()
{
    double expOutput = 0;
    for (int outputs = 0; outputs < neuronCounts[totalLayers - 1]; outputs++)
    {
        double activation = networkStructure[totalLayers - 1][outputs]->getActivation();
        expOutput = expOutput + exp(activation);
    }
    return expOutput;
}

double network::backPropagation()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    

}

void network::calcBackDerivToCost(int layer, int weightID, int neuronPos, node* neuron)
{
    
     node* nextNeuron = networkStructure[layer + 1][weightID];
     double score = neuron->getWeight(weightID) * derivativeSigmoid(nextNeuron->getRawForwardSum()) * deriv[weightID];
     atomicAdd(&deriv[neuronPos],score);
    
    
}

double network::derivativeSigmoid(double x)
{
    return(1 - sigmoid(x));
}

void network::calcReplacementDerivs(int layer)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int weightID = index; weightID < neuronCounts[layer + 1]; weightID = weightID + stride)
    {
        int neuron = weightID / neuronCounts[layer + 1];
        node* neuronObj = networkStructure[layer][neuron];
        calcBackDerivToCost(layer, weightID, neuron, neuronObj);
    }
}

void network::calcBackOneLayer(int layer)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int taskID = index; taskID < taskCount[layer]; taskID = taskID + stride)
    {
        int neuron = taskID / neuronCounts[layer];
        int weightID = taskID % neuronCounts[layer + 1];
        node* neuronObj = networkStructure[layer][neuron];
        node* targetObj = networkStructure[layer+1][neuron];
        double newDeriv = getSingleDeriv(weightID);
        if (neuron < neuronCounts[layer])
        {
            double weightAdd = neuronObj->getActivation() * derivativeSigmoid(targetObj->getRawForwardSum()) * newDeriv;
            neuronObj->updateWeightUpdate(weightAdd, weightID, 0);
        }
        else
        {
            double newBias = 1 * derivativeSigmoid(targetObj->getRawForwardSum()) * newDeriv;
            targetObj->updateBiasUpdates(0,newBias);
        }
      
    }
    
}

double network::getSingleDeriv(double newDeriv)
{
    return newDeriv;
}

void network::performWeightsandBiasUpdates()
{
    /*
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int taskID = index; taskID < totalTaskCount; taskID = taskID + stride)
    {
        for (int layer = 0; layer < totalLayers; layer++)
        {
            if (taskID < taskCount[layer])
            {

            }
        }
    }
    */
}


